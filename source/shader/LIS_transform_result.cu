#include "hip/hip_runtime.h"
#version 430
#extension GL_ARB_shading_language_420pack : require
#extension GL_NV_gpu_shader5 : require
#extension GL_ARB_explicit_attrib_location : require

//layout(local_size_x = 16) in;
layout(local_size_x = 32, local_size_y = 16) in;

layout(std430, binding = 0) buffer bit_array_buffer
{
	uint data[];
};

layout(std430, binding = 1) buffer bit_array_out_buffer
{
	uint data_out[];
};

layout(std430, binding = 2) buffer marker_vector_buffer
{
	vec2  marker_origin_distorted_interleaved[]; //4 vec2 marker orig 4 vec2 marker distort
};

layout(std430, binding = 3) buffer marker_aaba_buffer
{
	vec4	  aaba_buffer[];
};

layout(std430, binding = 4) buffer marker_transform_buffer
{
	vec4  marker_affine_transform_coeffs[]; //1 vec4 = a[4] 1 vec4 = b[4]
};

struct ssbo_uniform {
	uvec2 image_dimensions;
	uvec2 image_out_dimensions;
	uint image_byte_length;
	uint image_out_byte_length;
	uvec2 nbr_of_marker;
	float	   process_status;
};

layout(std430, binding = 5) buffer uniform_ssbo_buffer
{
	ssbo_uniform  ssbo_uniforms;
};


//uniform uvec2   image_dimensions;
//uniform uvec2   image_out_dimensions;
//uniform uint    image_byte_length;
//uniform uint    image_out_byte_length;
//
//uniform uvec2   nbr_of_marker;
//
uniform float	process_status;

const int nbr_of_points_per_marker_square = 8;

uint
get_sample_data(vec2 in_sampling_pos) {

	uvec2 image_dimensions = ssbo_uniforms.image_dimensions;
	uvec2 pixel_pos = uvec2(in_sampling_pos * image_dimensions);
	uint byte_pos = pixel_pos.x + image_dimensions.x * pixel_pos.y;

	uint byte_offset = byte_pos / 32;
	uint bit_offset = byte_pos % 32;

	//GLSL is working with 32bit/64bit values only
	//data storage is 32bit	
	//
	//get 0/1 from bit 
	uint value = 0;
	value = uint(data[byte_offset] >> bit_offset) & 0x000001;


	return value;
}

void
set_sample_data(uvec2 store_pos, uint value) {

	uvec2 image_out_dimensions = ssbo_uniforms.image_out_dimensions;

	uint byte_offset = (store_pos.x + store_pos.y *image_out_dimensions.x) / 32;
	uint bit_offset = (store_pos.x + store_pos.y * image_out_dimensions.x) % 32;


	//GLSL is working with 32bit/64bit values only
	//data storage is 32bit	
	//
	//set 0/1 from bit 
	atomicOr(data_out[byte_offset], (value << bit_offset));// uint(data_out[byte_offset] | (1 << bit_offset));
	
}


int
intersect(in vec2 point, in int marker_nbr, in int interleaved_lookup) {

	uint index = 0;

	bool quad_hit = false;

	uvec2 nbr_of_marker = ssbo_uniforms.nbr_of_marker;
	//uvec2 image_out_dimensions = ssbo_uniforms.image_out_dimensions;


	for (index = 0; index != (nbr_of_marker.x * nbr_of_marker.y); ++index) {

		int i, j = 0;
		bool c = false;
		const int nvert = 4;

		vec2 vert[nvert];

		vert[0] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 0 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];
		vert[1] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 1 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];
		vert[2] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 2 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];
		vert[3] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 3 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];

		//
		for (i = 0, j = nvert - 1; i < nvert; j = i++) {
			if (((vert[i].y > point.y) != (vert[j].y > point.y))
				&& (point.x < (vert[j].x - vert[i].x) * (point.y - vert[i].y) / (vert[j].y - vert[i].y) + vert[i].x)
				)
				c = !c;
		}

		if (c) {
			quad_hit = true;
			break;
		}
	}

	if (quad_hit)
		return int(index);
	else
		return -1;


}

uint
affine_transform_color(vec2 frag_uv, int index) {

	vec4 a = marker_affine_transform_coeffs[2u * index]; //+ 0
	vec4 b = marker_affine_transform_coeffs[2u * index + 1];

	vec2 src_cord = vec2(a[0] * frag_uv.x + a[1] * frag_uv.y + a[2] * frag_uv.x * frag_uv.y + a[3],
		b[0] * frag_uv.x + b[1] * frag_uv.y + b[2] * frag_uv.x * frag_uv.y + b[3]);

	return get_sample_data(src_cord);
}

shared uint result_shared_data[16];//32*16 / 32 = 16

void main()
{
	uint byte_shared_pos = gl_LocalInvocationID.y;// (gl_LocalInvocationID.x + gl_LocalInvocationID.y * 16) / 32;
	uint bit_shared_pos = gl_LocalInvocationID.x % 32;

	if (0 == bit_shared_pos) {
		result_shared_data[byte_shared_pos] = 0;
	}
	//barrier();

	uvec2 image_dimensions = ssbo_uniforms.image_dimensions;
	uvec2 image_out_dimensions = ssbo_uniforms.image_out_dimensions;
	
	float process_status = ssbo_uniforms.process_status;// uintBitsToFloat(ssbo_uniforms[4].x);
	float step_size_y = float(image_out_dimensions.y) / float(image_dimensions.y);

	//check square
	uvec2 storePos = gl_GlobalInvocationID.xy;
	
	vec2 floating_pos_global = vec2(gl_GlobalInvocationID.xy) / vec2(ssbo_uniforms.image_out_dimensions);
	floating_pos_global.y = step_size_y * float(gl_GlobalInvocationID.y)/float(ssbo_uniforms.image_out_dimensions.y) + process_status;
	vec2 floating_pos_local = vec2(gl_GlobalInvocationID.xy) / vec2(ssbo_uniforms.image_out_dimensions);

	int index = intersect(floating_pos_global, 0, 1);
	uint sample_data = affine_transform_color(floating_pos_global, index);

#if 1//global memory
	set_sample_data(storePos, sample_data);
#else //shared memory
	atomicOr(result_shared_data[byte_shared_pos], (sample_data << bit_shared_pos));// uint(data_out[byte_offset] | (1 << bit_offset));
	//result_shared_data[byte_shared_pos] = result_shared_data[byte_shared_pos] | (sample_data << bit_shared_pos);
	
//	barrier();
	if (0 == bit_shared_pos) {		
		uint byte_pos = (gl_GlobalInvocationID.x + gl_GlobalInvocationID.y * ssbo_uniforms.image_out_dimensions.x)/32;
		data_out[byte_pos] = result_shared_data[byte_shared_pos];
	}
#endif
}



