#include "hip/hip_runtime.h"
#version 430
#extension GL_ARB_shading_language_420pack : require
#extension GL_NV_gpu_shader5 : require
#extension GL_ARB_explicit_attrib_location : require


layout(std430, binding = 0) buffer bit_array_buffer
{
	uint data[];
};

layout(std430, binding = 1) buffer marker_vector_buffer
{
	vec2  marker_origin_distorted_interleaved[]; //4 vec2 marker orig 4 vec2 marker distort
};

layout(std430, binding = 2) buffer marker_aaba_buffer
{
	vec4	  aaba_buffer[];
};

layout(std430, binding = 3) buffer marker_transform_buffer
{
	vec4  marker_affine_transform_coeffs[]; //1 vec4 = a[4] 1 vec4 = b[4]
};


layout(location = 0) out vec4 FragColor;

uniform mat4 Modelview;

uniform uvec2   image_dimensions;
uniform uint    image_byte_length;

uniform uvec2   nbr_of_marker;

const int nbr_of_points_per_marker_square = 8;

vec4
get_sample_data(vec2 in_sampling_pos) {


	uvec2 pixel_pos = uvec2(in_sampling_pos * image_dimensions);
	uint byte_pos = pixel_pos.x + image_dimensions.x * pixel_pos.y;

	uint byte_offset = byte_pos / 32;
	uint bit_offset = byte_pos % 32;

	uint value = 0;
	value = uint(data[byte_offset] >> bit_offset) & 0x000001;


	return vec4(float(value), 0.0, 0.0, 1.0);

	return vec4(float(pixel_pos.x) / image_dimensions.x, float(pixel_pos.y) / image_dimensions.y, 0.0, 1.0);

}

bool
intersectPointInTriangle(in vec2 p, in vec2 p0, in vec2 p1, in vec2 p2)
{
	float s = p0.y * p2.x - p0.x * p2.y + (p2.y - p0.y) * p.x + (p0.x - p2.x) * p.y;
	float t = p0.x * p1.y - p0.y * p1.x + (p0.y - p1.y) * p.x + (p1.x - p0.x) * p.y;

	if ((s < 0) != (t < 0))
		return false;

	float A = -p1.y * p2.x + p0.y * (p2.x - p1.x) + p0.x * (p1.y - p2.y) + p1.x * p2.y;
	if (A < 0.0)
	{
		s = -s;
		t = -t;
		A = -A;
	}
	return s > 0 && t > 0 && (s + t) <= A;
}

int
intersect(in vec2 point, in int marker_nbr, in int interleaved_lookup) {

	bool aaba_found = false;
	uint index = 0;

	//check AABA first
	//for (int j = 0; j != nbr_of_marker.y; ++j) {
	//	if (aaba_found)
	//		break;
	//	for (int i = 0; i != nbr_of_marker.x; ++i) {
	//		index = i + j * nbr_of_marker.x;
	//		vec4 aaba = aaba_buffer[index];

	//		if (point.x > aaba.x 
	//			&& point.x < aaba.z
	//			&& point.y > aaba.y 
	//			&& point.y < aaba.w) {
	//			aaba_found = true;
	//			break;
	//		}
	//	}
	//}

	//if (!aaba_found)
	//	return -1;
	//else
	//	return int(index);

	bool quad_hit;

	for (index = 0; index != (nbr_of_marker.x * nbr_of_marker.y); ++index) {

		int i, j = 0;
		bool c = false;
		const int nvert = 4;

		vec2 vert[nvert];

		vert[0] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 0 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];
		vert[1] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 1 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];
		vert[2] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 2 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];
		vert[3] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 3 + (interleaved_lookup * nbr_of_points_per_marker_square / 2)];

		//
		for (i = 0, j = nvert - 1; i < nvert; j = i++) {
			if (((vert[i].y > point.y) != (vert[j].y > point.y))
				&& (point.x < (vert[j].x - vert[i].x) * (point.y - vert[i].y) / (vert[j].y - vert[i].y) + vert[i].x)
				)
				c = !c;
		}

		if (c) {
			quad_hit = true;
			break;
		}
	}

	if (quad_hit)
		return int(index);
	else
		return -1;


}

vec4
affine_transform_color(vec2 frag_uv, int index) {

	//	vec2 src[4];
	//
	//	src[0] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 0];
	//	src[1] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 1];
	//	src[2] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 2];
	//	src[3] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 3];
	//
	//	vec2 dst[4];
	//
	//	dst[0] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 0 + (nbr_of_points_per_marker_square / 2)];
	//	dst[1] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 1 + (nbr_of_points_per_marker_square / 2)];
	//	dst[2] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 2 + (nbr_of_points_per_marker_square / 2)];
	//	dst[3] = marker_origin_distorted_interleaved[(index * nbr_of_points_per_marker_square) + 3 + (nbr_of_points_per_marker_square / 2)];
	//
	//	//calc affine transform
	//	float x1 = dst[0].x;
	//	float x2 = dst[1].x;
	//	float x3 = dst[2].x;
	//	float x4 = dst[3].x;
	//
	//	float y1 = dst[0].y;
	//	float y2 = dst[1].y;
	//	float y3 = dst[2].y;
	//	float y4 = dst[3].y;
	//
	//	vec4 vxn = vec4(src[0].x, src[1].x, src[2].x, src[3].x);
	//	vec4 vyn = vec4(src[0].y, src[1].y, src[2].y, src[3].y);
	//	/*mat4 M = mat4(x1, y1, x1*y1, 1,
	//				  x2, y2, x2*y2, 1,
	//				  x3, y3, x3*y3, 1,
	//				  x4, y4, x4*y4, 1	
	//	);
	//*/
	//	mat4 M = mat4(x1, x2, x3, x4,
	//				  y1, y2, y3, y4,
	//				  x1*y1, x2*y2, x3*y3, x4*y4,
	//				  1, 1, 1, 1
	//				);
	//
	//
	//
	//	mat4 inverseM = inverse(M);
	//
	//	//a = M^ * x;
	//	vec4 a = inverseM * vxn;
	//	vec4 b = inverseM * vyn;

	vec4 a = marker_affine_transform_coeffs[2u * index]; //+ 0
	vec4 b = marker_affine_transform_coeffs[2u * index + 1];

	vec2 src_cord = vec2(a[0] * frag_uv.x + a[1] * frag_uv.y + a[2] * frag_uv.x * frag_uv.y + a[3],
		b[0] * frag_uv.x + b[1] * frag_uv.y + b[2] * frag_uv.x * frag_uv.y + b[3]);

	//return vec4(src_cord, 0.0, 0.0);
	return get_sample_data(src_cord);
}

void main()
{

	/// Init color of fragment
	//vec4 dst = get_sample_data(frag_uv);
	vec4 dst = vec4(0.0);
	//check square
	int index = intersect(frag_uv, 0, 1);

	if (index != -1) {

		vec4 sample_data = affine_transform_color(frag_uv, index);

		//dst += /*vec4(0.0, 0.2, 0.0, 0.0) +*/ abs(sample_data - vec4(frag_uv, 0.0, 0.0));
		dst = /*vec4(0.0, 0.1, 0.0, 0.0) + */sample_data;
	}



	//index = intersect(frag_uv, 0, 0);
	//if (index != -1)
	//	dst += vec4(0.0, 0.0, 0.1, 0.0);

	FragColor = dst;
}
